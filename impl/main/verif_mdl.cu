#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

/*static float _pourcent_masque_nulle[C] = {0};
static float _alpha[C] = {0.01};

__global__
static void kerd_p1e5(float * p, uint i, float _1E5) {
	p[i] += _1E5;
};

static void p1e5(Mdl_t * mdl, uint c, uint i, float _1E5, uint _MODE) {
	if (_MODE == 0) {
		mdl->p[c][i] += _1E5;
	} else {
		kerd_p1e5<<<1,1>>>(mdl->p__d[c], i, _1E5);
		ATTENDRE_CUDA();
	}
};*/

static void __performance() {
	/*ASSERT(C == 11);
	titre("Performance");
	//
	uint Y[C] = {
		512,
		256,
		256,
		256,
		128,
		64,
		32,
		16,
		8,
		4,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		LSTM1D,
		LSTM1D,
		LSTM1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	uint lignes[BLOQUES] = {0};
	FOR(0, i, BLOQUES) lignes[i] = rand() % EMA_INTS;
	Mdl_t * mdl = cree_mdl(Y, insts, lignes);
	plumer_mdl(mdl);
	//
	uint plus_T = 16*16*25;
	//
	mdl_plume_grad(mdl, DEPART, DEPART+plus_T);
	//
	printf("TEMPS MODEL = ");
	MESURER(mdl_aller_retour(mdl, DEPART, DEPART+plus_T, 3));
	//
	liberer_mdl(mdl);*/
};

static void __verif_mdl_1e5() {
	/*ASSERT(C == 3);
	titre("Comparer MODEL 1e-5");
	//
	uint Y[C] = {
		64,
		16,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		DOT1D,
		DOT1D
	};
	ema_int_t * bloque[BLOQUES] = {
	//			    Source,      Nature,  K_ema, Intervalle, decale,     {params}
		cree_ligne(SRC_PRIXS,    DIRECTE,   1,       5,        2,      cree_DIRECTE()),
		cree_ligne(SRC_VOLULMES, MACD,      300,     10,       0,      cree_MACD(1)),
		cree_ligne(SRC_PRIXS,    DIRECTE,   1,       1,        12,     cree_DIRECTE()),
		cree_ligne(SRC_HIGH,     CHIFFRE,   12,      32,       8,      cree_CHIFFRE(500)),
		cree_ligne(SRC_LOW,      DXDX,      1,       1,        0,      cree_DXDX(12, 20, 12, 7)),
		cree_ligne(SRC_PRIXS,    DX,        50,      250,      0,      cree_DX(7, 15)),
		cree_ligne(SRC_VOLULMES, MACD,       2,      37,       7,      cree_MACD(7)),
		cree_ligne(SRC_HIGH,     MACD,      20,      4,        28,     cree_MACD(21))
	};
	Mdl_t * mdl = cree_mdl(Y, insts, bloque);
	plumer_mdl(mdl);
	//
	uint plus_T = 16*16*1;
	//
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, 16*16);
	//
	//mdl_plume_poids(mdl);
	//
	//comportement(mdl, DEPART, DEPART+16*16);
#define MODE 3 //0,1,2,3
	//
	MESURER(mdl_aller_retour(mdl, DEPART, DEPART+plus_T, MODE));
	mdl_gpu_vers_cpu(mdl);
	//
	//	1e-5
	//
	mdl_zero_gpu(mdl);
	float _f = mdl_score(mdl, DEPART, DEPART+plus_T, MODE);
	float _1E5 = 1e-3;
	FOR(0, c, C) {
		printf("###############################################################\n");
		printf("                       C = %2.i (%s)    \n", c, nom_inst[mdl->insts[c]]);
		printf("#######################vvvvvvvvvvvvvv##########################\n");
		//
		FOR(0, i, mdl->inst_POIDS[c]) {
			p1e5(mdl, c, i, +_1E5, MODE);
			float grad_1e5 = (mdl_score(mdl, DEPART, DEPART+plus_T, MODE) - _f)/_1E5;
			p1e5(mdl, c, i, -_1E5, MODE);
			//
			float a=grad_1e5, b=mdl->dp[c][i];
			printf("%i| ", i);
			PLUME_CMP(a, b);
			printf("\n");
		}
	};
	printf("  1e5 === df(x)  \n");

	//
	liberer_mdl(mdl);*/
};

void verif_mdl_1e5() {
	__performance();
	__verif_mdl_1e5();
};