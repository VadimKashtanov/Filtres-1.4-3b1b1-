#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

/*
Utilisation :
	./prog2__resultats mdl.bin
*/

void __gain(Mdl_t * mdl, uint t0) {
	uint t1 = t0 + 14*24;	//336h % 16 == 0
	printf("gain t0=%i t1=%i [2 semaines] = %f$\n", t0, t1, mdl_gain(mdl, t0, t1, 3));
};

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	float * ancien = mdl_pred(mdl, t0, t1, 3);
	printf("PRED %i %i = ", t0, t1);
	FOR(0, p, P) printf("%f%% ", 100*ancien[p]);
	printf("\n");
	free(ancien);
};

int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");  charger_tout();
	//
	if (argc == 2) {
		Mdl_t * mdl = ouvrire_mdl(argv[1]);
		mdl_verif(mdl);
		//
		uint t0 = DEPART;
		uint t1 = ROND_MODULO(FIN, 16);
		//
		plume_pred(mdl, t0, t1);
		plume_pred(mdl, t0, t0+16*1000);
		plume_pred(mdl, t0+16*1000, t1-16*1000);
		plume_pred(mdl, t1-16*1000, t1);
		plume_pred(mdl, t1-16*100, t1);
		//
		titre("Gain en 2 semaines");
		__gain(mdl, t0);
		__gain(mdl, t0+5000);
		__gain(mdl, t0+10000);
		__gain(mdl, t1-2*365*24);
		__gain(mdl, t1-356*24);
		__gain(mdl, t1-3*30*24);
		__gain(mdl, t1-14*24);
		//
		liberer_mdl(mdl);
	} else {
		ERR("./prog2__resultats mdl.bin")
	}
	liberer_tout();
}