#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	float * ancien = mdl_pred(mdl, t0, t1, 3);
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*ancien[p]);
	printf("\n");
	free(ancien);
};

float pourcent_masque_nulle[C] = {0};

float * pourcent_masque = de_a(0.10, 0.00, C);

//	# Un jour reflechire a f(x@p0 + b0) * f(x@p1 + b1) + f(x@p2 + b2)

float * alpha = /*de_a(1e-2, 1e-2, C);//*/de_a(1e-2, 1e-4, C);

//	## (x/3) * (x-2)**2                     ##
//	## score(x) + rnd()*abs(score(x))*0.05  ##

uint optimiser_tous_les__nulle[C] = UNIFORME_C(1);

uint optimiser_tous_les[C] = UNIFORME_C(1);/*{
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1,
	1
};*/

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	//
	//pourcent_masque[0] /= 10;
	//pourcent_masque[1] /= 10;
	alpha[0] *= 100;
	//
	MSG("S(x) Ajouter un peut d'aléatoire");
	MSG("S(x) Eventuellement faire des prediction plus lointaines");
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	//titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");
	ecrire_structure_generale("structure_generale.bin");

	uint Y[C] = {
		1024,
		512,
		256,
		512,
		256,
		128,
		256,
		128,
		64,
		128,
		64,
		32,
		64,
		32,
		16,
		32,
		16,
		8,
		16,
		8,
		4,
		8,
		4,
		2,
		4,
		2,
		P
	};
	uint insts[C] = UNIFORME_C(DOT1D);//{
	insts[0] = FILTRES_PRIXS;
	//
	//	Assurances :
	ema_int_t * bloque[BLOQUES] = {
	//			    Source,      Nature,  K_ema, Intervalle, decale,     {params}
	// ----
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 256, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 32, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 1.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 8, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 2.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 16, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 8.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 64, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 16.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 128, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 32.0, 0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 256, 0, cree_DIRECTE()),
	// ----
		cree_ligne(SRC_PRIXS, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 128, 128, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 1, 1, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 4, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 1.0, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 16, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 4.0, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 64, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 128, 8.0, 0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 128, 128, 0, cree_MACD(1)),
	// ----
		cree_ligne(SRC_HIGH, CHIFFRE, 1, 1, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 8, 8, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 32, 32, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 128, 128, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 1, 1, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 8, 8, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 32, 32, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_HIGH, CHIFFRE, 128, 128, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 1, 1, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 8, 8, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 32, 32, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 128, 128, 0, cree_CHIFFRE(1000)),
		cree_ligne(SRC_LOW, CHIFFRE, 1, 1, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 8, 8, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 32, 32, 0, cree_CHIFFRE(10000)),
		cree_ligne(SRC_LOW, CHIFFRE, 128, 128, 0, cree_CHIFFRE(10000))
	};
	//
	Mdl_t * mdl = cree_mdl(Y, insts, bloque);

	//Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	enregistrer_les_lignes_brute(mdl, "lignes_brute.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	plume_pred(mdl, t0, t1);
	//comportement(mdl, t0, t0+16*16);
	//
	srand(time(NULL));
#define PERTURBATIONS 0
	//
	uint REP = 300;
	FOR(0, rep, REP) {
		perturber(mdl, 10);
		perturber_filtres(mdl, 100);
		optimisation_mini_packet(
			mdl,
			t0, t1, 16*16*1,
			alpha, 1.0,
			RMSPROP, 40,
			pourcent_masque,
			PERTURBATIONS,
			optimiser_tous_les);
		/*optimiser(
			mdl,
			t0, t1,
			alpha, 1.0,
			RMSPROP, 150,
			//pourcent_masque_nulle);
			pourcent_masque,
			PERTURBATIONS,
			optimiser_tous_les);*/
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		plume_pred(mdl, t0, t1);
		//
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, REP);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};