#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static float filtre(uint depart, float * x, float * f, uint intervalle, uint decale) {
	float normer_x[N];
	//
	float _min=x[depart - (decale+0)*intervalle], _max=x[depart - (decale+1)*intervalle];
	normer_x[0] = _min;
	FOR(1, i, N) {
		float a = x[depart - (decale+i)*intervalle];
		normer_x[i] = a;
		if (a > _max) _max = a;
		if (a < _min) _min = a;
	}
	FOR(0, i, N) normer_x[i] = (normer_x[i]-_min)/(_max-_min);
	//
	float s = 0, d = 0;
	float f_nouveau = f[0];
	float x_nouveau = normer_x[0];
	s += sqrtf(1 + fabs(x_nouveau - f_nouveau));
	float f_avant = f_nouveau;
	float x_avant = x_nouveau;
	FOR(1, i, N) {
		f_nouveau = f[i];
		x_nouveau = normer_x[i];
		s += sqrtf(1 + fabs(  x_nouveau   -   f_nouveau  ));
		d += powf((1 + fabs((x_nouveau-x_avant) - (f_nouveau-f_avant))), 2);
		f_avant   = f_nouveau;
		x_avant   = x_nouveau;
	};

	s = s/8-1;
	d = d/7-1;

	return 2*expf(-s*s -d*d)-1;
};


int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	//
	FILE * fp = fopen(argv[1], "rb");
	//
	uint Y[C];
	FREAD(Y, sizeof(uint), C, fp);
	//
	uint PRIXS_bitget;
	FREAD(&PRIXS_bitget, sizeof(uint), 1, fp);
	uint intervalles[BLOQUES], decales[BLOQUES];
	FREAD(intervalles, sizeof(uint), BLOQUES, fp);
	FREAD(decales,     sizeof(uint), BLOQUES, fp);
	//
	float * lignes = alloc<float>(PRIXS_bitget*BLOQUES);
	FREAD(lignes, sizeof(float), PRIXS_bitget*BLOQUES, fp);
	//
	float * poids[C];
	FOR(0, c, C) {
		uint POIDS;
		FREAD(&POIDS, sizeof(uint), 1, fp);
		poids[c] = alloc<float>(POIDS);
		FREAD(poids[c], sizeof(float), POIDS, fp);
	}
	//
	fclose(fp);

	//	------------- Calcule ----------------
	float * y_avant   = alloc<float>( PRIXS_bitget*MAX_Y );
	float * y_nouveau = alloc<float>( PRIXS_bitget*MAX_Y );
	//
	/*#pragma omp parallel
	#pragma omp for*/
	FOR(0, f, BLOQUES*F_PAR_BLOQUES) {
		uint b = (f - (f % F_PAR_BLOQUES)) / F_PAR_BLOQUES;
		FOR(DEPART, t, PRIXS_bitget) {
			y_nouveau[t*MAX_Y + f] = filtre(
				b*PRIXS_bitget + t,
				lignes,
				poids[0] + f*N,
				intervalles[b], decales[b]
			);
		}
	};
	/*#pragma omp parallel
	#pragma omp for*/
	FOR(0, i, PRIXS_bitget*MAX_Y) y_avant[i] = y_nouveau[i];
	//
	FOR(1, c, C) {
		uint X = Y[c-1];
		/*#pragma omp parallel
		#pragma omp for*/
		FOR(0, i, Y[c]) {
			FOR(DEPART, t, PRIXS_bitget) {
				float s = poids[c][(X+1)*i + X-1+1];
				FOR(0, j, X) s += poids[c][(X+1)*i + j] * y_avant[t*MAX_Y + j];
				y_nouveau[t*MAX_Y + i] = tanh(s);
			};
		};

		/*#pragma omp parallel
		#pragma omp for*/
		FOR(0, i, PRIXS_bitget*MAX_Y) y_avant[i] = y_nouveau[i];
	};

	//	---------- Ecrire Resultat ----------
	fp = fopen(argv[1], "wb");
	float res[PRIXS_bitget];
	FOR(DEPART, t, PRIXS_bitget) res[t] = y_nouveau[t*MAX_Y + 0];
	FWRITE(res+DEPART, sizeof(float), (PRIXS_bitget-DEPART), fp);
	fclose(fp);
}