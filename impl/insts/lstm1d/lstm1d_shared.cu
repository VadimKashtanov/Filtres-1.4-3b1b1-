#include "hip/hip_runtime.h"
#include "lstm1d.cuh"

//(0) Ft[t] = logistic(x@Wf + Ct[t-1]@Uf + Bf)
//(1) It[t] = logistic(x@Wi + Ct[t-1]@Ui + Bi)
//(2) Ot[t] = logistic(x@Wo + Ct[t-1]@Uo + Bo)
//(3) Tt[t] = tanh    (x@Wt + Bt)
//(4) Ct[t] = Ft[t]*Ct[-1] + It[t]*Tt[t]
//(5) Ht[t] = Ot[t]*Ct[t]

#define BLOQUE_Y 8

#define K16 64

static __global__ void kerd_lstm1d__shared__0123(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	float * _x = x + t*X_vars + DEPART_x;
	//
	__shared__ float sF[BLOQUE_Y], sI[BLOQUE_Y], sO[BLOQUE_Y], sT[BLOQUE_Y];
	if (thy == 0) {
		sF[thx] = 0;
		sI[thx] = 0;
		sO[thx] = 0;
		sT[thx] = 0;
	}
	__syncthreads();
	//
	uint k;
	if (_y < Y) {
		//	--- x@W ---
		FOR(0, _k, X/K16) {
			k = _k*K16 + thy;
			float __x = _x[k];
			atomicAdd(&sF[thx], __x * p[Wf+_y*X+k]);
			atomicAdd(&sI[thx], __x * p[Wi+_y*X+k]);
			atomicAdd(&sO[thx], __x * p[Wo+_y*X+k]);
			atomicAdd(&sT[thx], __x * p[Wt+_y*X+k]);
		}
		//	--- Ct[t-1]@U ---
		FOR(0, _k, Y/K16) {
			k = _k*K16 + thy;
			float __c = y[Ct-Y_vars+k];
			atomicAdd(&sF[thx], __c * p[Uf+_y*Y+k]);
			atomicAdd(&sI[thx], __c * p[Ui+_y*Y+k]);
			atomicAdd(&sO[thx], __c * p[Uo+_y*Y+k]);
		}
		__syncthreads();

		if (thy == 0) {
			y[Ft+_y] = sF[thx];
			y[It+_y] = sI[thx];
			y[Ot+_y] = sO[thx];
			y[Tt+_y] = sT[thx];
		}
	}
};

static __global__ void kerd_lstm1d__shared__45(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (_y < Y) {
		float ft=y[Ft+_y]+p[Bf+_y], it=y[It+_y]+p[Bi+_y], ot=y[Ot+_y]+p[Bo+_y], tt=y[Tt+_y]+p[Bt+_y];
		//
		ft = 1 / (1 + expf(-ft));
		it = 1 / (1 + expf(-it));
		ot = 1 / (1 + expf(-ot));
		tt = tanh(tt);
		//
		float __Ct = ft*y[Ct-Y_vars+_y] + it*tt;
		y[Ct+_y] = __Ct;
		y[Ht+_y] = __Ct * ot;
	}
};

void nvidia_lstm1d__shared(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	ASSERT(X  % K16 == 0);
	ASSERT(Y  % K16 == 0);
	kerd_lstm1d__shared__0123<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y, K16)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
	kerd_lstm1d__shared__45<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
};

//	===========================================================================
//	===========================================================================
//	===========================================================================

static __global__ void deriv_kerd_lstm1d__shared__45(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (_y < Y) {
		/*float ft=y[Ft+_y]+p[Bf+_y], it=y[It+_y]+p[Bi+_y], ot=y[Ot+_y]+p[Bo+_y], tt=y[Tt+_y]+p[Bt+_y];
		//
		ft = 1 / (1 + expf(-ft));
		it = 1 / (1 + expf(-it));
		ot = 1 / (1 + expf(-ot));
		tt = tanh(tt);
		//
		float __Ct = ft*y[Ct-Y_vars+_y] + it*tt;
		y[Ct+_y] = __Ct;
		y[Ht+_y] = __Ct * ot;*/
		float ft=y[Ft+_y], it=y[It+_y], ot=y[Ot+_y], tt=y[Tt+_y];
		float __Ct = y[Ct+_y];
		float __Ct1 = y[Ct-Y_vars+_y];
		float dht = dy[Ht+_y];
		float d__Ct = dht * ot;
		float d__ot = dht * __Ct;
		//
		float d__ft = d__Ct * __Ct1;
		//float d__Ct1 = d__Ct * ft;
		float d__it = d__Ct * tt;
		float d__tt = d__Ct * it;
		dy[Ft+_y] += d__ft;
		dy[It+_y] += d__it;
		dy[Ot+_y] += d__ot;
		dy[Tt+_y] += d__tt;
		//
		float dsF = d__ft * (ft*(1-ft));
		float dsI = d__it * (it*(1-it));
		float dsO = d__ot * (ot*(1-ot));
		float dsT = d__tt * (1-tt*tt);
		//
		p[Bf+_y] += dsF;
		p[Bi+_y] += dsI;
		p[Bo+_y] += dsO;
		p[Bt+_y] += dsT;
	}
};

static __global__ void deriv_kerd_lstm1d__shared__0123(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	float * _x = x + t*X_vars + DEPART_x;
	float * _dx = dx + t*X_vars + DEPART_x;
	//
	__shared__ float dsF[BLOQUE_Y], dsI[BLOQUE_Y], dsO[BLOQUE_Y], dsT[BLOQUE_Y];
	//
	uint k;
	if (_y < Y) {
		if (thy == 0) {
			float ft=y[Ft+_y], it=y[It+_y], ot=y[Ot+_y], tt=y[Tt+_y];
			//
			float d__ft = dy[Ft+_y];
			float d__it = dy[It+_y];
			float d__ot = dy[Ot+_y];
			float d__tt = dy[Tt+_y];
			//
			dsF[thx] = d__ft * (ft*(1-ft));
			dsI[thx] = d__it * (it*(1-it));
			dsO[thx] = d__ot * (ot*(1-ot));
			dsT[thx] = d__tt * (1-tt*tt);
		}
		__syncthreads();
		//	--- x@W ---
		FOR(0, _k, X/K16) {
			k = _k*K16 + thy;
			/*float __x = _x[k];
			atomicAdd(&sF[thx], __x * p[Wf+_y*X+k]);
			atomicAdd(&sI[thx], __x * p[Wi+_y*X+k]);
			atomicAdd(&sO[thx], __x * p[Wo+_y*X+k]);
			atomicAdd(&sT[thx], __x * p[Wt+_y*X+k]);*/
			float d__x = 0;
			float __x = _x[k];
	//		sF += __x * Wf[_y*X+k];
			d__x += dsF[thx] * p[Wf+_y*X+k];
			atomicAdd(&dp[Wf+_y*X+k], dsF[thx] * __x);
	//		sI += __x * Wi[_y*X+k];
			d__x += dsI[thx] * p[Wi+_y*X+k];
			atomicAdd(&dp[Wi+_y*X+k], dsI[thx] * __x);
	//		sO += __x * Wo[_y*X+k];
			d__x += dsO[thx] * p[Wo+_y*X+k];
			atomicAdd(&dp[Wo+_y*X+k], dsO[thx] * __x);
	//		sT += __x * Wt[_y*X+k];
			d__x += dsT[thx] * p[Wt+_y*X+k];
			atomicAdd(&dp[Wt+_y*X+k], dsT[thx] * __x);
			//
			atomicAdd(&_dx[k], d__x);
		}
		//	--- Ct[t-1]@U ---
		FOR(0, _k, Y/K16) {
			k = _k*K16 + thy;
			/*float __c = y[Ct-Y_vars+k];
			atomicAdd(&sF[thx], __c * p[Uf+_y*Y+k]);
			atomicAdd(&sI[thx], __c * p[Ui+_y*Y+k]);
			atomicAdd(&sO[thx], __c * p[Uo+_y*Y+k]);*/
			//
			float d__c = 0;
			float __c = y[Ct-Y_vars+k];
	//		sF += __c * Uf[_y*Y+k];
			d__c += dsF[thx] * p[Uf+_y*Y+k];
			atomicAdd(&dp[Uf+_y*Y+k], dsF[thx] * __c);
	//		sI += __c * Ui[_y*Y+k];
			d__c += dsI[thx] * p[Ui+_y*Y+k];
			atomicAdd(&dp[Ui+_y*Y+k], dsI[thx] * __c);
	//		sO += __c * Uo[_y*Y+k];
			d__c += dsO[thx] * p[Uo+_y*Y+k];
			atomicAdd(&dp[Uo+_y*Y+k], dsO[thx] * __c);
			//
			atomicAdd(&dy[Ct-Y_vars+k], d__c);
		}
		//__syncthreads();
	}
};

void d_nvidia_lstm1d__shared(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	ASSERT(X  % K16 == 0);
	ASSERT(Y  % K16 == 0);
	deriv_kerd_lstm1d__shared__45<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
	deriv_kerd_lstm1d__shared__0123<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y, K16)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};