#include "hip/hip_runtime.h"
#include "lstm1d.cuh"

//(0) Ft[t] = logistic(x@Wf + Ct[t-1]@Uf + Bf)
//(1) It[t] = logistic(x@Wi + Ct[t-1]@Ui + Bi)
//(2) Ot[t] = logistic(x@Wo + Ct[t-1]@Uo + Bo)
//(3) Tt[t] = tanh    (x@Wt + Bt)
//(4) Ct[t] = Ft[t]*Ct[-1] + It[t]*Tt[t]
//(5) Ht[t] = Ot[t]*Ct[t]

#define BLOQUE_Y 32

static __global__ void kerd_lstm1d_naive__0123(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;

	float * _x = x + t*X_vars + DEPART_x;

	if (_y < Y) {
		float sF = p[Bf+_y];
		float sI = p[Bi+_y];
		float sO = p[Bo+_y];
		float sT = p[Bt+_y];
		// --- x@W ---
		FOR(0, k, X) {
			float __x = _x[k];
			sF += __x * p[Wf+_y*X+k];
			sI += __x * p[Wi+_y*X+k];
			sO += __x * p[Wo+_y*X+k];
			sT += __x * p[Wt+_y*X+k];
		}
		// --- Ct[t-1]@U ---
		FOR(0, k, Y) {
			float __c = y[Ct-Y_vars+k];
			sF += __c * p[Uf+_y*Y+k];
			sI += __c * p[Ui+_y*Y+k];
			sO += __c * p[Uo+_y*Y+k];
		}
		// --- logistic && tanh ---
		y[Ft+_y] = 1 / (1 + expf(-sF));
		y[It+_y] = 1 / (1 + expf(-sI));
		y[Ot+_y] = 1 / (1 + expf(-sO));
		y[Tt+_y] = tanh(sT);
	}
};

static __global__ void kerd_lstm1d_naive__45(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;

	if (_y < Y) {
		y[Ct+_y] = y[Ft+_y] * y[Ct-Y_vars+_y] + y[It+_y]*y[Tt+_y];
		y[Ht+_y] = y[Ct+_y] * y[Ot+_y];
	}
};

void nvidia_lstm1d_naive(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	kerd_lstm1d_naive__0123<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
	kerd_lstm1d_naive__45<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
};

//	===========================================================================
//	===========================================================================
//	===========================================================================

static __global__ void deriv_kerd_lstm1d_naive__45(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;

	if (_y < Y) {
		//Ht[_y] = Ct[_y] * Ot[_y];
		dy[Ct+_y] += dy[Ht+_y] * y[Ot+_y];
		dy[Ot+_y] += dy[Ht+_y] * y[Ct+_y];

		//Ct[_y] = Ft[_y]*Ct[_y - 1*(6*Y)] + It[_y]*Tt[_y];
		dy[Ft +_y]  += dy[Ct+_y] * y[Ct-Y_vars+_y];
		dy[Ct-Y_vars+_y] += dy[Ct+_y] * y[Ft+_y];
		//
		dy[It +_y] += dy[Ct+_y] * y[Tt+_y];
		dy[Tt +_y] += dy[Ct+_y] * y[It+_y];
	}
};

static __global__ void deriv_kerd_lstm1d_naive__0123(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;

	float *  _x =  x + t*X_vars + DEPART_x;
	float * _dx = dx + t*X_vars + DEPART_x;

	if (_y < Y) {
		float dsF = dy[Ft+_y] * (y[Ft+_y] * (1 - y[Ft+_y]));
		float dsI = dy[It+_y] * (y[It+_y] * (1 - y[It+_y]));
		float dsO = dy[Ot+_y] * (y[Ot+_y] * (1 - y[Ot+_y]));
		float dsT = dy[Tt+_y] * (  1 - y[Tt+_y]*y[Tt+_y]  );
		//	--- Ct[t-1]@U ---
		FOR(0, k, Y) {
			float d__c = 0;
			float __c = y[Ct-Y_vars+k];
	//		sF += __c * Uf[_y*Y+k];
			d__c += dsF * p[Uf+_y*Y+k];
			atomicAdd(&dp[Uf+_y*Y+k], dsF * __c);
	//		sI += __c * Ui[_y*Y+k];
			d__c += dsI * p[Ui+_y*Y+k];
			atomicAdd(&dp[Ui+_y*Y+k], dsI * __c);
	//		sO += __c * Uo[_y*Y+k];
			d__c += dsO * p[Uo+_y*Y+k];
			atomicAdd(&dp[Uo+_y*Y+k], dsO * __c);
			//
			atomicAdd(&dy[Ct-Y_vars+k], d__c);
		}
		//	--- x@W ---
		FOR(0, k, X) {
			float d__x = 0;
			float __x = _x[k];
	//		sF += __x * Wf[_y*X+k];
			d__x += dsF * p[Wf+_y*X+k];
			atomicAdd(&dp[Wf+_y*X+k], dsF * __x);
	//		sI += __x * Wi[_y*X+k];
			d__x += dsI * p[Wi+_y*X+k];
			atomicAdd(&dp[Wi+_y*X+k], dsI * __x);
	//		sO += __x * Wo[_y*X+k];
			d__x += dsO * p[Wo+_y*X+k];
			atomicAdd(&dp[Wo+_y*X+k], dsO * __x);
	//		sT += __x * Wt[_y*X+k];
			d__x += dsT * p[Wt+_y*X+k];
			atomicAdd(&dp[Wt+_y*X+k], dsT * __x);
			//
			atomicAdd(&_dx[k], d__x);
		}
		//
	//	float sF=Bf[_y], sI=Bi[_y], sO=Bo[_y], sT=Bt[_y];
		atomicAdd(&dp[Bf+_y], dsF);
		atomicAdd(&dp[Bi+_y], dsI);
		atomicAdd(&dp[Bo+_y], dsO);
		atomicAdd(&dp[Bt+_y], dsT);
	}
};

void d_nvidia_lstm1d_naive(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	deriv_kerd_lstm1d_naive__45<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
	deriv_kerd_lstm1d_naive__0123<<<dim3(KERD(Y, BLOQUE_Y)), dim3(BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		t,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};