#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../../impl_tmpl/tmpl_etc.cu"

#define pseudo_alea_d_une_grain(i) ((float)((121+(i%1234))*31 % 1001 ) / 1001.0)

//	===============================================================

static __global__ void kerd_nvidia_score_somme(
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	float s = 0;
	FOR(0, i, T) {
		FOR(0, p, P) {
			y[(depart + i)*P + p] += pseudo_alea_d_une_grain(i + ((uint)y[(depart+i)*P+p] % 10001)) * SCORE_Y_COEF_BRUIT;
			s += (P-p)*cuda_SCORE(
				y[(depart+i)*P+p], _PRIXS[depart+i+p+1], _PRIXS[depart+i/*+p*/]
			);
		}
	}
	*score = s;
};

float nvidia_somme_score(float * y, uint depart, uint T)
{
	float * somme_score__d = cudalloc<float>(1);
	kerd_nvidia_score_somme<<<1,1>>>(
		y, depart, T,
		somme_score__d, prixs__d
	);
	ATTENDRE_CUDA();
	float somme_score;
	CONTROLE_CUDA(hipMemcpy(&somme_score, somme_score__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipFree(somme_score__d));
	return (somme_score);
};

float  nvidia_score_finale(float somme, uint T) {
	return APRES_SCORE(somme / (float)(P * T));
};

//	===============================================================

float d_nvidia_score_finale(float somme, uint T) {
	return dAPRES_SCORE(somme / (float)(P * T));
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	float _dy, float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		FOR(0, p, P) {
			dy[(depart+_t)*P+p] = _dy * (P-p)*cuda_dSCORE(
				y[(depart+_t)*P+p], _PRIXS[depart+_t+p+1], _PRIXS[depart+_t/*+p*/]
			) / ((float)T*P);
		}
	}
};

void d_nvidia_somme_score(float d_score, float * y, float * dy, uint depart, uint T) {
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		d_score,
		y, dy,
		depart, T,
		prixs__d
	);
	ATTENDRE_CUDA();
};