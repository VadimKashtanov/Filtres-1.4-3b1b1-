#include "hip/hip_runtime.h"
#include "lstm1d.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_lstm1d(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	mdl->inst_POIDS        [c] = 3*(X*Y + Y*Y + Y) + 1*(X*Y + Y);
	mdl->inst_VARS         [c] = mdl->Y[c]*6;	//Ft, It, Ot, Tt, Ct, Ht
	mdl->inst_LOCDS        [c] = 0;
	mdl->inst_SORTIES      [c] = mdl->Y[c];		//Ht
	mdl->inst_DEPART_SORTIE[c] = mdl->inst_VARS[c] - mdl->inst_SORTIES[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);
	FOR(0, i, mdl->inst_POIDS[c])
		mdl->p[c][i] = (2*rnd()-1) * sqrtf(2.0 / (float)X);
};

void plume_lstm1d(Mdl_t * mdl, uint c)
{
	printf("POIDS LSTM: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	//
	float * p = mdl->p[c];
	//
	printf("Wf : "); FOR(0, i, X*Y) {printf("%+f,", p[Wf+i]);}; printf("\n");
	printf("Wi : "); FOR(0, i, X*Y) {printf("%+f,", p[Wi+i]);}; printf("\n");
	printf("Wo : "); FOR(0, i, X*Y) {printf("%+f,", p[Wo+i]);}; printf("\n");
	//
	printf("Uf : "); FOR(0, i, Y*Y) {printf("%+f,", p[Uf+i]);}; printf("\n");
	printf("Ui : "); FOR(0, i, Y*Y) {printf("%+f,", p[Ui+i]);}; printf("\n");
	printf("Uo : "); FOR(0, i, Y*Y) {printf("%+f,", p[Uo+i]);}; printf("\n");
	//
	printf("Bf : "); FOR(0, i,   Y) {printf("%+f,", p[Bf+i]);}; printf("\n");
	printf("Bi : "); FOR(0, i,   Y) {printf("%+f,", p[Bi+i]);}; printf("\n");
	printf("Bo : "); FOR(0, i,   Y) {printf("%+f,", p[Bo+i]);}; printf("\n");
	//
	printf("Wt : "); FOR(0, i, X*Y) {printf("%+f,", p[Wt+i]);}; printf("\n");
	printf("Bt : "); FOR(0, i,   Y) {printf("%+f,", p[Bt+i]);}; printf("\n");
};

void intel_lstm1d(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	float * _x = x + t*X_vars + DEPART_x;
	//(0) Ft[t] = logistic(x@Wf + Ct[t-1]@Uf + Bf)
	//(1) It[t] = logistic(x@Wi + Ct[t-1]@Ui + Bi)
	//(2) Ot[t] = logistic(x@Wo + Ct[t-1]@Uo + Bo)
	//(3) Tt[t] = tanh    (x@Wt + Bt)
	//(4) Ct[t] = Ft[t]*Ct[-1] + It[t]*Tt[t]
	//(5) Ht[t] = Ot[t]*Ct[t]
	FOR(0, _y, Y) {
		float sF=p[Bf+_y];
		float sI=p[Bi+_y];
		float sO=p[Bo+_y];
		float sT=p[Bt+_y];
		//	--- x@W ---
		FOR(0, k, X) {
			float __x = _x[k];
			sF += __x * p[Wf+_y*X+k];
			sI += __x * p[Wi+_y*X+k];
			sO += __x * p[Wo+_y*X+k];
			sT += __x * p[Wt+_y*X+k];
		}
		//	--- Ct[t-1]@U ---
		FOR(0, k, Y) {
			float __c = y[Ct-Y_vars+k];
			sF += __c * p[Uf+_y*Y+k];
			sI += __c * p[Ui+_y*Y+k];
			sO += __c * p[Uo+_y*Y+k];
		}
		//	--- logistic && tanh ---
		y[Ft+_y] = 1 / (1 + expf(-sF));
		y[It+_y] = 1 / (1 + expf(-sI));
		y[Ot+_y] = 1 / (1 + expf(-sO));
		y[Tt+_y] = tanh(sT);
	}
	FOR(0, _y, Y) {
		y[Ct+_y] = y[Ft+_y] * y[Ct-1*Y_vars+_y] + y[It+_y]*y[Tt+_y];
		y[Ht+_y] = y[Ct+_y] * y[Ot+_y];
	};
}

void d_intel_lstm1d(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	float * _x  = x  + t*X_vars + DEPART_x;
	float * _dx = dx + t*X_vars + DEPART_x;

	FOR(0, _y, Y) {
		dy[Ct+_y] += y[Ot+_y] * dy[Ht+_y];
		dy[Ot+_y] += y[Ct+_y] * dy[Ht+_y];
		//
		dy[Ft+_y] += y[Ct-1*Y_vars+_y] * dy[Ct+_y];
		dy[Ct-1*Y_vars+_y] += y[Ft+_y] * dy[Ct+_y];
		dy[It+_y] += y[Tt+_y] * dy[Ct+_y];
		dy[Tt+_y] += y[It+_y] * dy[Ct+_y];
	}

	FOR(0, _y, Y) {
		//	--- logistic && tanh ---
	//	Ft[_y] = 1 / (1 + expf(-sF));
	//	It[_y] = 1 / (1 + expf(-sI));
	//	Ot[_y] = 1 / (1 + expf(-sO));
	//	Tt[_y] = tanh(sT);
		//
		float dsF = dy[Ft+_y] * (y[Ft+_y] * (1 - y[Ft+_y]));
		float dsI = dy[It+_y] * (y[It+_y] * (1 - y[It+_y]));
		float dsO = dy[Ot+_y] * (y[Ot+_y] * (1 - y[Ot+_y]));
		float dsT = dy[Tt+_y] * (  1 - y[Tt+_y]*y[Tt+_y]  );

		//	--- Ct[t-1]@U ---
		FOR(0, k, Y) {
			float d__c = 0;
			float __c = y[Ct-1*Y_vars+k];	//t-1
	//		sF += __c * Uf[_y*Y+k];
			d__c += dsF * p[Uf+_y*Y+k];
			dp[Uf+_y*Y+k] += dsF * __c;
	//		sI += __c * Ui[_y*Y+k];
			d__c += dsI * p[Ui+_y*Y+k];
			dp[Ui+_y*Y+k] += dsI * __c;
	//		sO += __c * Uo[_y*Y+k];
			d__c += dsO * p[Uo+_y*Y+k];
			dp[Uo+_y*Y+k] += dsO * __c;
			//
			dy[Ct-1*Y_vars+k] += d__c;
		}
		//	--- x@W ---
		FOR(0, k, X) {
	//		float __x = _x[k];
			float d__x = 0;
			float __x = _x[k];	//t-1
	//		sF += __x * Wf[_y*X+k];
			d__x += dsF * p[Wf+_y*X+k];
			dp[Wf+_y*X+k] += dsF * __x;
	//		sI += __x * Wi[_y*X+k];
			d__x += dsI * p[Wi+_y*X+k];
			dp[Wi+_y*X+k] += dsI * __x;
	//		sO += __x * Wo[_y*X+k];
			d__x += dsO * p[Wo+_y*X+k];
			dp[Wo+_y*X+k] += dsO * __x;
	//		sT += __x * Wt[_y*X+k];
			d__x += dsT * p[Wt+_y*X+k];
			dp[Wt+_y*X+k] += dsT * __x;
			//
			_dx[k] += d__x;
		}
		//
	//	float sF=Bf[_y], sI=Bi[_y], sO=Bo[_y], sT=Bt[_y];
		dp[Bf+_y] += dsF;
		dp[Bi+_y] += dsI;
		dp[Bo+_y] += dsO;
		dp[Bt+_y] += dsT;
	}
}

//	=========================================================
__global__
static void kerd_cuda_memset_t(float * v, uint t, uint vars) {
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < vars) {
		v[t*vars + thx] = 0;
	}
};
void cuda_memset_t(float * v, uint t, uint vars) {
	kerd_cuda_memset_t<<<dim3(KERD(vars,32)), dim3(32)>>>(v, t, vars);
	ATTENDRE_CUDA();
}

void f_lstm1d(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	//
	if (mode == 0) {
		memset(
			mdl->y[inst]+(t0-1)*mdl->inst_VARS[inst],
			0,
			sizeof(float)*mdl->inst_VARS[inst]
		);
		FOR(t0, t, t1) {
			intel_lstm1d(
				X_vars, Y_vars,
				X, Y,
				t,
				DEPART_x,
				mdl->y[inst-1], mdl->y[inst],
				mdl->p[inst],
				mdl->l[inst]);
		}
	} else if (mode == 1) {
		cuda_memset_t(
			mdl->y__d[inst],
			t0-1, mdl->inst_VARS[inst]
		);
		FOR(t0, t, t1) {
			nvidia_lstm1d_naive(
				X_vars, Y_vars,
				X, Y,
				t,
				DEPART_x,
				mdl->y__d[inst-1], mdl->y__d[inst],
				mdl->p__d[inst],
				mdl->l__d[inst]);
		}
	}  else if (mode == 2 || mode == 3) {
		cuda_memset_t(
			mdl->y__d[inst],
			t0-1,
			mdl->inst_VARS[inst]
		);
		FOR(t0, t, t1) {
			nvidia_lstm1d__shared(
				X_vars, Y_vars,
				X, Y,
				t,
				DEPART_x,
				mdl->y__d[inst-1], mdl->y__d[inst],
				mdl->p__d[inst],
				mdl->l__d[inst]);
		}
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_lstm1d(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	//
	if (mode == 0) {
		RETRO_FOR(t0, t, t1) {
			d_intel_lstm1d(
				X_vars, Y_vars,
				X, Y,
				t,
				DEPART_x,
				mdl->y[inst-1], mdl->y[inst],
				mdl->p[inst],
				mdl->l[inst],
				mdl->dy[inst],
				mdl->dy[inst-1],
				mdl->dp[inst]);
		}
	} else if (mode == 1) {
		RETRO_FOR(t0, t, t1) {
			d_nvidia_lstm1d_naive(
				X_vars, Y_vars,
				X, Y,
				t,
				DEPART_x,
				mdl->y__d[inst-1], mdl->y__d[inst],
				mdl->p__d[inst],
				mdl->l__d[inst],
				mdl->dy__d[inst],
				mdl->dy__d[inst-1],
				mdl->dp__d[inst]);
		}
	}  else if (mode == 2 || mode == 3) {
		RETRO_FOR(t0, t, t1) {
			d_nvidia_lstm1d__shared(
				X_vars, Y_vars,
				X, Y,
				t,
				DEPART_x,
				mdl->y__d[inst-1], mdl->y__d[inst],
				mdl->p__d[inst],
				mdl->l__d[inst],
				mdl->dy__d[inst],
				mdl->dy__d[inst-1],
				mdl->dp__d[inst]);
		}
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}