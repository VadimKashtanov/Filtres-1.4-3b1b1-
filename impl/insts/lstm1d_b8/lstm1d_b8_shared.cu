#include "hip/hip_runtime.h"
#include "lstm1d_b8.cuh"

//(0) Ft[t] = logistic(x@Wf + Ct[t-1]@Uf + Bf)
//(1) It[t] = logistic(x@Wi + Ct[t-1]@Ui + Bi)
//(2) Ot[t] = logistic(x@Wo + Ct[t-1]@Uo + Bo)
//(3) Tt[t] = tanh    (x@Wt + Bt)
//(4) Ct[t] = Ft[t]*Ct[-1] + It[t]*Tt[t]
//(5) Ht[t] = Ot[t]*Ct[t]

#define BLOQUE_Y 8

#define K16 64

/*static __global__ void kerd_lstm1d_b8__shared__0123(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	
};

static __global__ void kerd_lstm1d_b8__shared__45(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	
};*/

void nvidia_lstm1d_b8__shared(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	TODO()
};

//	===========================================================================
//	===========================================================================
//	===========================================================================

/*static __global__ void deriv_kerd_lstm1d_b8__shared__45(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	
};

static __global__ void deriv_kerd_lstm1d_b8__shared__0123(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	
};*/

void d_nvidia_lstm1d_b8__shared(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint t,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	TODO()
};